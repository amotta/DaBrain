#include <stdio.h>
#include <hip/hip_runtime.h>
#include "gpu.h"

void gpuCopyMemory(const void * hPtr, void ** dPtr, size_t size){
	hipError_t error = hipSuccess;

	error = hipMalloc(dPtr, size);
	if(error != hipSuccess){
		printf("Failed to allocate device memory\n");
		return;
	}

	error = hipMemcpy((void *) *dPtr, hPtr, size, hipMemcpyHostToDevice);
	if(error != hipSuccess){
		printf("Failed to copy data. Error:\n");
		printf("%s\n", hipGetErrorString(error));
		return;
	}
}