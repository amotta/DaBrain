#include <stdio.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "gpu.h"

static bool ready = false;
static hipblasHandle_t handle;

void gpuInit(){
	hipblasStatus_t status = hipblasCreate(&handle);
	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Failed to create cuBLAS handle\n");
		return;
	}

	ready = true;
}

void gpuCopyMemory(const void * hPtr, void ** dPtr, size_t size){
	hipError_t error = hipSuccess;

	error = hipMalloc(dPtr, size);
	if(error != hipSuccess){
		printf("Failed to allocate device memory\n");
		return;
	}

	error = hipMemcpy((void *) *dPtr, hPtr, size, hipMemcpyHostToDevice);
	if(error != hipSuccess){
		printf("Failed to copy data. Error:\n");
		printf("%s\n", hipGetErrorString(error));
		return;
	}
}

void gpuMultiplyMV(
	const float * mat,
	const float * vecIn,
	float * vecOut,
	int rows, int cols
){
	if(!ready){
		gpuInit();
	}

	const float alpha = 1.0f;
	const float beta = 0.0f;

	hipblasStatus_t status;
	status = hipblasSgemv(
		handle,
		// no transformation
		HIPBLAS_OP_N,
		// dimensions of S
		rows, cols,
		// only product (alpha = 1)
		&alpha,
		// synapse matrix
		mat,
		// leading dimension of synapse matrix
		rows, 
		// vector
		vecIn,
		// stride between elements
		1,
		// no addition (beta = 0)
		&beta,
		// result
		vecOut,
		// stride between elements
		1
	);

	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Error in matrix vector multiplication\n");
		return;
	}
}