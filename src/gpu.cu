#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "gpu.h"
#include "neuron.h"

static bool ready = false;
static hipblasHandle_t handle;

void gpuInit(){
	hipblasStatus_t status = hipblasCreate(&handle);
	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Failed to create cuBLAS handle\n");
		return;
	}

	ready = true;
}

void gpuCopyMemory(const void * hPtr, void ** dPtr, size_t size){
	hipError_t error = hipSuccess;

	error = hipMalloc(dPtr, size);
	if(error != hipSuccess){
		printf("Failed to allocate device memory\n");
		return;
	}

	error = hipMemcpy((void *) *dPtr, hPtr, size, hipMemcpyHostToDevice);
	if(error != hipSuccess){
		printf("Failed to copy data. Error:\n");
		printf("%s\n", hipGetErrorString(error));
		return;
	}
}

void gpuMultiplyMV(
	const float * mat,
	int matRows,
	int matCols,
	const float * vecIn,
	int vecInStride,
	float * vecOut,
	int vecOutStride
){
	if(!ready){
		gpuInit();
	}

	const float alpha = 1.0f;
	const float beta = 0.0f;

	hipblasStatus_t status;
	status = hipblasSgemv(
		handle,
		// no transformation
		HIPBLAS_OP_N,
		// dimensions of S
		matRows, matCols,
		// only product (alpha = 1)
		&alpha,
		// synapse matrix
		mat,
		// leading dimension of synapse matrix
		matRows, 
		// vector
		vecIn,
		// stride between elements
		vecInStride,
		// no addition (beta = 0)
		&beta,
		// result
		vecOut,
		// stride between elements
		vecOutStride
	);

	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Error in matrix vector multiplication\n");
		return;
	}
}

__global__ void updateState(
	float * dynState,
	const float * dynParam
){
	// neuron id
	int nId = blockDim.x * blockIdx.x + threadIdx.x;

	// pointer to corresponding column
	float * nDynState = &dynState[DYN_STATE_LEN * nId];
	const float * nDynParam = &dynParam[DYN_PARAM_LEN * nId];

	float v = nDynState[DYN_STATE_V];
	float u = nDynState[DYN_STATE_U];
	float I = nDynState[DYN_STATE_I_SYN];

	if(v >= 30.0f){
		v = nDynParam[DYN_PARAM_C];
		u = u + nDynParam[DYN_PARAM_D];

		// neuron is firing
		nDynState[DYN_STATE_FIRING] = 1.0f;
	}else{
		// not firing
		nDynState[DYN_STATE_FIRING] = 0.0f;
	}

	// update state
	v += 0.5f * (0.04f * v * v + 5.0f * v + 140 - u + I + 5.0f);
	v += 0.5f * (0.04f * v * v + 5.0f * v + 140 - u + I + 5.0f);
	u += nDynParam[DYN_PARAM_A] * (nDynParam[DYN_PARAM_B] * v - u);

	// write result
	dynState[DYN_STATE_V] = v;
	dynState[DYN_STATE_U] = u;
}

#define BLOCK_SIZE (32 * 32)
void gpuUpdateState(
	int numNeurons,
	float * dynState,
	const float * dynParam
){
	dim3 threads(BLOCK_SIZE);
	dim3 grid((int) ceil((double) numNeurons / BLOCK_SIZE));

	updateState<<<grid, threads>>>(dynState, dynParam);
}
