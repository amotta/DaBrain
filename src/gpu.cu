#include <math.h>
#include <stdio.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "gpu.h"
#include "neuron.h"

static bool ready = false;
static hipblasHandle_t handle;

void gpuInit(){
	hipblasStatus_t status = hipblasCreate(&handle);
	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Failed to create cuBLAS handle\n");
		return;
	}

	ready = true;
}

void gpuCopyMemoryToGPU(const void * hPtr, void ** dPtr, size_t size){
	hipError_t error = hipSuccess;

	error = hipMalloc(dPtr, size);
	if(error != hipSuccess){
		printf("Failed to allocate device memory\n");
		return;
	}

	error = hipMemcpy((void *) *dPtr, hPtr, size, hipMemcpyHostToDevice);
	if(error != hipSuccess){
		printf("Failed to copy data to GPU. Error:\n");
		printf("%s\n", hipGetErrorString(error));
		return;
	}
}

void gpuCopyMemoryFromGPU(const void * dPtr, void * hPtr, size_t size){
	hipError_t error = hipSuccess;

	error = hipMemcpy(hPtr, dPtr, size, hipMemcpyDeviceToHost);
	if(error != hipSuccess){
		printf("Failed to copy data to host. Error:\n");
		printf("%s\n", hipGetErrorString(error));
		return;
	}
}

int gpuMultiplyBMV(
	const float * mat,
	int matRows,
	int matCols,
	int matSuper,
	int matSub,
	const float * vecIn,
	int vecInStride,
	float * vecOut,
	int vecOutStride
){
	if(!ready){
		gpuInit();
	}

	const float alpha = 1.0f;
	const float beta = 0.0f;

	hipblasStatus_t status;
	status = hipblasSgbmv(
		handle,
		// no transformation
		HIPBLAS_OP_N,
		// matrix dimensions
		matRows, matCols,
		// lower and upper diagonals
		matSub, matSuper,
		// alpha
		&alpha,
		// matrix
		mat,
		// leading dimension of matrix
		matSuper + 1 + matSub,
		// vector
		vecIn,
		// vector stride
		vecInStride,
		// beta
		&beta,
		// output
		vecOut,
		// output stride
		vecOutStride
	);

	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Error in banded matrix vector multiplication.\n");
		return -1;
	}

	return 0;
}

int gpuMultiplyMV(
	const float * mat,
	int matRows,
	int matCols,
	const float * vecIn,
	int vecInStride,
	float * vecOut,
	int vecOutStride
){
	if(!ready){
		gpuInit();
	}

	const float alpha = 1.0f;
	const float beta = 0.0f;

	hipblasStatus_t status;
	status = hipblasSgemv(
		handle,
		// no transformation
		HIPBLAS_OP_N,
		// dimensions of S
		matRows, matCols,
		// only product (alpha = 1)
		&alpha,
		// synapse matrix
		mat,
		// leading dimension of synapse matrix
		matRows, 
		// vector
		vecIn,
		// stride between elements
		vecInStride,
		// no addition (beta = 0)
		&beta,
		// result
		vecOut,
		// stride between elements
		vecOutStride
	);

	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Error in matrix vector multiplication\n");
		return -1;
	}

	return 0;
}

