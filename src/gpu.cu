#include <stdio.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "gpu.h"

static hipblasHandle_t handle;

int gpuInit(){
	// we prefer L1 cache
	hipError_t error;
	error = hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	if(error != hipSuccess){
		printf("Could not set cache config\n");
		return -1;
	}

	// init cuBLAS library
	hipblasStatus_t status = hipblasCreate(&handle);
	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Failed to create cuBLAS handle\n");
		return -1;
	}

	return 0;
}

int gpuCopyTo(
	const size_t size,
	const void * hPtr,
	void ** dPtr
){
	hipError_t error;

	// allocate memory on GPU
	error = hipMalloc(dPtr, size);

	if(error != hipSuccess){
		printf("Failed to allocate device memory\n");
		return -1;
	}

	// copy data to GPU
	error = hipMemcpy(
		(void *) *dPtr,
		hPtr,
		size,
		hipMemcpyHostToDevice
	);

	if(error != hipSuccess){
		printf("Failed to copy data to GPU. Error:\n");
		printf("%s\n", hipGetErrorString(error));
		return -1;
	}

	return 0;
}

int gpuCopyFrom(
	const size_t size,
	const void * dPtr,
	void * hPtr
){
	hipError_t error = hipSuccess;

	// copy data from GPU
	error = hipMemcpy(
		hPtr,
		dPtr,
		size,
		hipMemcpyDeviceToHost
	);

	if(error != hipSuccess){
		printf("Failed to copy data to host. Error:\n");
		printf("%s\n", hipGetErrorString(error));
		return -1;
	}

	return 0;
}

int gpuMultiplyBMV(
	const float * mat,
	int matRows,
	int matCols,
	int matSuper,
	int matSub,
	const float * vecIn,
	int vecInStride,
	float * vecOut,
	int vecOutStride
){
	const float alpha = 1.0f;
	const float beta = 0.0f;

	hipblasStatus_t status;
	status = hipblasSgbmv(
		handle,
		// no transformation
		HIPBLAS_OP_N,
		// matrix dimensions
		matRows, matCols,
		// lower and upper diagonals
		matSub, matSuper,
		// alpha
		&alpha,
		// matrix
		mat,
		// leading dimension of matrix
		matSuper + 1 + matSub,
		// vector
		vecIn,
		// vector stride
		vecInStride,
		// beta
		&beta,
		// output
		vecOut,
		// output stride
		vecOutStride
	);

	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Error in banded matrix vector multiplication.\n");
		return -1;
	}

	return 0;
}

int gpuMultiplySV(
	int vecRows,
	const float * alpha,
	float * vec
){
	hipblasStatus_t status;
	status = hipblasSscal(
		handle,
		// vector size
		vecRows,
		// scaling factor
		alpha,
		// vector
		vec,
		// stride between elements
		1
	);

	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Error in vector scaling\n");
		return -1;
	}

	return 0;
}
