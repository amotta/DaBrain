#include <math.h>
#include <stdio.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "gpu.h"
#include "neuron.h"

static hipblasHandle_t handle;

int  gpuInit(){
	// we prefer L1 cache
	hipError_t error;
	error = hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	if(error != hipSuccess){
		printf("Could not set cache config\n");
		return -1;
	}

	// init cuBLAS library
	hipblasStatus_t status = hipblasCreate(&handle);
	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Failed to create cuBLAS handle\n");
		return -1;
	}

	return 0;
}

void gpuCopyMemoryToGPU(const void * hPtr, void ** dPtr, size_t size){
	hipError_t error = hipSuccess;

	error = hipMalloc(dPtr, size);
	if(error != hipSuccess){
		printf("Failed to allocate device memory\n");
		return;
	}

	error = hipMemcpy((void *) *dPtr, hPtr, size, hipMemcpyHostToDevice);
	if(error != hipSuccess){
		printf("Failed to copy data to GPU. Error:\n");
		printf("%s\n", hipGetErrorString(error));
		return;
	}
}

void gpuCopyMemoryFromGPU(const void * dPtr, void * hPtr, size_t size){
	hipError_t error = hipSuccess;

	error = hipMemcpy(hPtr, dPtr, size, hipMemcpyDeviceToHost);
	if(error != hipSuccess){
		printf("Failed to copy data to host. Error:\n");
		printf("%s\n", hipGetErrorString(error));
		return;
	}
}

int gpuMultiplyBMV(
	const float * mat,
	int matRows,
	int matCols,
	int matSuper,
	int matSub,
	const float * vecIn,
	int vecInStride,
	float * vecOut,
	int vecOutStride
){
	const float alpha = 1.0f;
	const float beta = 0.0f;

	hipblasStatus_t status;
	status = hipblasSgbmv(
		handle,
		// no transformation
		HIPBLAS_OP_N,
		// matrix dimensions
		matRows, matCols,
		// lower and upper diagonals
		matSub, matSuper,
		// alpha
		&alpha,
		// matrix
		mat,
		// leading dimension of matrix
		matSuper + 1 + matSub,
		// vector
		vecIn,
		// vector stride
		vecInStride,
		// beta
		&beta,
		// output
		vecOut,
		// output stride
		vecOutStride
	);

	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Error in banded matrix vector multiplication.\n");
		return -1;
	}

	return 0;
}

int gpuMultiplyMV(
	const float * mat,
	int matRows,
	int matCols,
	const float * vecIn,
	int vecInStride,
	float * vecOut,
	int vecOutStride
){
	const float alpha = 1.0f;
	const float beta = 0.0f;

	hipblasStatus_t status;
	status = hipblasSgemv(
		handle,
		// no transformation
		HIPBLAS_OP_N,
		// dimensions of S
		matRows, matCols,
		// only product (alpha = 1)
		&alpha,
		// synapse matrix
		mat,
		// leading dimension of synapse matrix
		matRows, 
		// vector
		vecIn,
		// stride between elements
		vecInStride,
		// no addition (beta = 0)
		&beta,
		// result
		vecOut,
		// stride between elements
		vecOutStride
	);

	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Error in matrix vector multiplication\n");
		return -1;
	}

	return 0;
}

int gpuMultiplySV(
	int vecRows,
	const float * alpha,
	float * vec
){
	hipblasStatus_t status;
	status = hipblasSscal(
		handle,
		// vector size
		vecRows,
		// scaling factor
		alpha,
		// vector
		vec,
		// stride between elements
		1
	);

	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Error in vector scaling\n");
		return -1;
	}

	return 0;
}
