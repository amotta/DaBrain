#include "hip/hip_runtime.h"
#include <stdio.h>
#include "izhikevich.h"

__global__ void izhikevichUpdateCUDA(
	int numNeurons,
	float * dynState,
	float * firing,
	const float * dynParam,
	const float * Isyn
){
	// neuron id
	int nId = blockDim.x * blockIdx.x + threadIdx.x;

	// let's not exaggerate
	if(nId >= numNeurons) return;

	// pointer to corresponding column
	float * nDynState = &dynState[DYN_STATE_LEN * nId];
	const float * nDynParam = &dynParam[DYN_PARAM_LEN * nId];

	float v = nDynState[DYN_STATE_V];
	float u = nDynState[DYN_STATE_U];
	// synaptic current + thalamic input
	float I = Isyn[nId] + 5.0f;

	if(v >= 30.0f){
		v = nDynParam[DYN_PARAM_C];
		u = u + nDynParam[DYN_PARAM_D];

		// neuron is firing
		firing[nId] = 1.0f;
	}else{
		// not firing
		firing[nId] = 0.0f;
	}

	// update state
	v += 0.5f * (0.04f * v * v + 5.0f * v + 140 - u + I);
	v += 0.5f * (0.04f * v * v + 5.0f * v + 140 - u + I);
	u += nDynParam[DYN_PARAM_A] * (nDynParam[DYN_PARAM_B] * v - u);

	// write result
	nDynState[DYN_STATE_V] = v;
	nDynState[DYN_STATE_U] = u;
}

#define NUM_WARPS 32
int izhikevichUpdateState(
	int numNeurons,
	float * dynState,
	float * firing,
	const float * dynParam,
	const float * Isyn
){
	// reset CUDA error
	hipGetLastError();

	// update neurons
	dim3 threads(32 * NUM_WARPS);
	dim3 grid((int) ceil((double) numNeurons / (32 * NUM_WARPS)));
	izhikevichUpdateCUDA<<<grid, threads>>>(
		numNeurons,
		dynState,
		firing,
		dynParam,
		Isyn
	);

	// check for error
	hipError_t error = hipGetLastError();
	if(error != hipSuccess){
		printf("Could not update neuron states. Error:\n");
		printf("%s", hipGetErrorString(error));
		return -1;
	}

	return 0;
}
